#include "hip/hip_runtime.h"

// __global__ is the function specifier to define the cuda kernal that run the actual GPU
__global__ void add(int *a, int *b, int *c){                                     
  int i = threadIdx.x + blockDim.x * blockIdx.x;                              // to add 2 vectors or arrays together
  c[i] = a[i] + b[i];
}
__managed__ int vector_a[256], vector_b[256], vector_c[256];                 // __managed__ is used for cuda to accessed from both CPU and device GPU

int main(){
  for(int i = 0; i < 256; i++){
    vector_a[i] = i;
    vector_b[i] = 256 - i;
  }

  // <<<1,256>>> represents the <<<block,threads per block>>> are used to run this code in parallel 
  add<<<1,256>>>(vector_a, vector_b, vector_c);                                

  hipDeviceSynchronize();                                        // will pass the execution of thos code and wait for it to complete on the GPU

  int result_sum = 0;
  for(i = 0; i < 256; i++){
    result_sum += vector_c[i];                                   // result to store the output. 
  }

  printf("result: sum = %d", result_sum);
}